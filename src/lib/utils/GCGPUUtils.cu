/*
 * This file is subject to the terms and conditions defined in
 * file 'LICENSE.txt', which is part of this source code package.
 */

#include "utils/GCGPUUtils.cuh"

bool cudaCheckError()
{
	const hipError_t hipError_t = hipGetLastError();
	if (hipError_t != 0)
	{
		std::cerr << "CUDA Error: " << hipGetErrorString(hipError_t)
		          << std::endl;
		return false;
	}
	return true;
}

size_t getDeviceInfo(bool verbose)
{
	int devicesNb = 0;
	hipGetDeviceCount(&devicesNb);
	cudaCheckError();
	std::cout << "\n"
	          << "*** GPUs INFORMATION ***"
	          << "\n"
	          << std::endl;
	std::cout << "Number of devices detected: " << devicesNb << std::endl;
	size_t freeMem, totalMem;
	int gpu_id_toUse = 0;
	size_t maxDeviceMem = 0;
	for (int d_id = 0; d_id < devicesNb; d_id++)
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, d_id);
		hipSetDevice(d_id);
		hipMemGetInfo(&freeMem, &totalMem);
		if (verbose)
		{
			std::cout << "Device name: " << deviceProp.name << std::endl;
			std::cout << "Compute capability: " << deviceProp.major << "."
			          << deviceProp.minor << std::endl;
			std::cout << "Number of asynchronous engines: "
			          << deviceProp.asyncEngineCount << std::endl;
			std::cout << "Device memory - Total memory: "
			          << totalMem / (1024 * 1024 * 1024)
			          << "GiB - Available memory: "
			          << freeMem / (1024 * 1024 * 1024) << "GiB \n"
			          << std::endl;
		}
		if (freeMem > maxDeviceMem)
		{
			maxDeviceMem = freeMem;
			gpu_id_toUse = d_id;
		}
	}
	std::cout << "Selected device id: " << gpu_id_toUse << "\n" << std::endl;
	hipSetDevice(gpu_id_toUse);
	return maxDeviceMem;
}

void gpuAssert(hipError_t code, const char* file, int line, bool abort)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
		        line);
		if (abort)
			exit(code);
	}
}
