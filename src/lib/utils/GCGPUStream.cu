/*
 * This file is subject to the terms and conditions defined in
 * file 'LICENSE.txt', which is part of this source code package.
 */

#include "utils/GCGPUStream.cuh"

GCGPUStream::GCGPUStream(unsigned int flags)
{
	hipStreamCreateWithFlags(&m_stream, flags);
}

const hipStream_t& GCGPUStream::getStream() const
{
	return m_stream;
}

GCGPUStream::~GCGPUStream()
{
	hipStreamDestroy(m_stream);
}
